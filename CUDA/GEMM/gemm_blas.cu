#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "../common/polybenchUtilFuncts.h"

#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i ,j , ld) ((( j )*( ld ))+( i ))

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size */
#define NI 4096
#define NJ 4096
#define NK 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 32412.0f
#define BETA 2123.0f

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void gemm(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i,j,k;
	
	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NJ; j++)
    	{
			C[i*NJ + j] *= BETA;
	
			for (k = 0; k < NK; ++k)
			{
	  			C[i*NJ + j] += ALPHA * A[i*NK + k] * B[k*NJ + j];
			}
      	}
	}
}


void init(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i, j;

  	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NK; j++)
		{
      		A[i*NK + j] = ((DATA_TYPE) i*j) / NI;
		}
	}

  	for (i = 0; i < NK; i++)
	{
    	for (j = 0; j < NJ; j++)
		{
      		B[i*NJ + j] = ((DATA_TYPE) i*j) / NJ;
		}
	}

  	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NJ; j++)
		{
      		C[i*NJ + j] = ((DATA_TYPE) i*j) / NJ;
		}
	}
}


void compareResults(DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
	int i, j, fail;
	fail = 0;
	
	// Compare C1 and C2
	for (i=0; i < NI; i++) 
	{
		for (j=0; j < NJ; j++) 
		{
			if (percentDiff(C[i*NJ + j], C_outputFromGpu[i*NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gemm_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{	
		c[i * NJ + j] *= BETA;
		int k;
		for(k=0; k < NK; k++)
		{
			c[i * NJ + j] += ALPHA * a[i * NK + k] * b[k * NJ +j];
		}
	}
}


void gemmCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
	double t_start, t_end;
	double t_start_k, t_end_k;

	hipblasStatus_t stat;
	hipblasHandle_t handle;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	DATA_TYPE alpha = ALPHA;
	DATA_TYPE beta = BETA;

	stat = hipblasCreate(&handle);
	
	t_start = rtclock();

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);

	
	t_start_k = rtclock();

	stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,  NI, NJ, NK, &alpha, 
		A_gpu, NK, B_gpu, NJ, &beta, 
		C_gpu, NI);
	if(stat != HIPBLAS_STATUS_SUCCESS){
		printf("Error in culbas sgemv 1, error code = %d\n", stat);
		return;
	}

	hipDeviceSynchronize();
	t_end_k = rtclock();


	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);    

	t_end = rtclock();
	fprintf(stdout, "cBLAS kernel : %0.6lf\n", t_end_k - t_start_k);
	fprintf(stdout, "cBLAS copy + kernel : %0.6lf\n", t_end - t_start);
	hipblasDestroy(handle);
	
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}
	

int main(int argc, char *argv[])
{

	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* C;  
	DATA_TYPE* C_outputFromGpu; 

	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE)); 
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));   
	C = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE)); 
	C_outputFromGpu = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE)); 

	init(A, B, C);
	
	GPU_argv_init();
	
	gemmCuda(A, B, C, C_outputFromGpu);


        /* 
	double t_start, t_end;
	t_start = rtclock();	
	gemm(A, B, C);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(C, C_outputFromGpu);
        */
	

	free(A);
	free(B);  
	free(C);  
	free(C_outputFromGpu); 

    	return 0;
}

