#include "hip/hip_runtime.h"
/**
 * mvt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../common/polybenchUtilFuncts.h"

#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDX2C(i ,j , ld) ((( j )*( ld ))+( i ))

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define N 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;


void init_array(DATA_TYPE* A, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
	int i, j;

	for (i = 0; i < N; i++)
	{
		x1[i] = ((DATA_TYPE) i) / N;
		x2[i] = ((DATA_TYPE) i + 1) / N;
		y1[i] = ((DATA_TYPE) i + 3) / N;
		y2[i] = ((DATA_TYPE) i + 4) / N;
		for (j = 0; j < N; j++)
		{
			A[i*N + j] = ((DATA_TYPE) i*j) / N;
		}
	}
}



void runMvt(DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
	int i, j;
	
	for (i=0; i<N; i++) 
	{
		for (j=0; j<N; j++) 
		{
       			x1[i] = x1[i] + a[i*N + j] * y1[j];
        	}
    	}
	
	for (i=0; i<N; i++) 
	{
		for (j=0; j<N; j++) 
		{
 		       	x2[i] = x2[i] + a[j*N + i] * y2[j];
      		}
    	}
}


void compareResults(DATA_TYPE* x1, DATA_TYPE* x1_outputFromGpu, DATA_TYPE* x2, DATA_TYPE* x2_outputFromGpu)
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<N; i++) 
	{
		if (percentDiff(x1[i], x1_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}

		if (percentDiff(x2[i], x2_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void mvt_kernel1(DATA_TYPE *a, DATA_TYPE *x1, DATA_TYPE *y_1)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j=0; j < N; j++)
		{
			x1[i] += a[i * N + j] * y_1[j];
		}
	}
}


__global__ void mvt_kernel2(DATA_TYPE *a, DATA_TYPE *x2, DATA_TYPE *y_2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j=0; j < N; j++)
		{
			x2[i] += a[j * N + i] * y_2[j];	
		}
	}
}

void mvtCuda(DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y_1, DATA_TYPE* y_2, 
			DATA_TYPE* x1_outputFromGpu, DATA_TYPE* x2_outputFromGpu)
{
	double t_start, t_end;
	double t_start_k, t_end_k;

	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	DATA_TYPE one = 1.0;
	DATA_TYPE zero = 0.0;

	DATA_TYPE* a_gpu;
	DATA_TYPE* x1_gpu;
	DATA_TYPE* x2_gpu;
	DATA_TYPE* y_1_gpu;
	DATA_TYPE* y_2_gpu;

	stat = hipblasCreate(&handle);

	t_start = rtclock();
	hipMalloc((void **)&a_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&x1_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&x2_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&y_1_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&y_2_gpu, sizeof(DATA_TYPE) * N);
	hipMemcpy(a_gpu, a, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(x1_gpu, x1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(x2_gpu, x2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_1_gpu, y_1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_2_gpu, y_2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	
	

	t_start_k = rtclock();
	stat = hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &one, a_gpu, 
		N, y_1_gpu, 1, &one, x1_gpu, 1);

	stat = hipblasSgemv(handle, HIPBLAS_OP_T, N, N, &one, a_gpu, 
		N, y_2_gpu, 1, &one, x2_gpu, 1);

	hipDeviceSynchronize();
	t_end_k = rtclock();


	hipMemcpy(x1_outputFromGpu, x1_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);
	hipMemcpy(x2_outputFromGpu, x2_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);    
	
	t_end = rtclock();

	fprintf(stdout, "cBLAS kernel : %0.6lfs\n", t_end_k - t_start_k);
	fprintf(stdout, "cBLAS data + kerenl : %0.6lfs\n", t_end - t_start);
	hipblasDestroy(handle);
	hipFree(a_gpu);
	hipFree(x1_gpu);
	hipFree(x2_gpu);
	hipFree(y_1_gpu);
	hipFree(y_2_gpu);
}


int main()
{
	double t_start, t_end;

	DATA_TYPE* a;
	DATA_TYPE* x1;
	DATA_TYPE* x2;
	DATA_TYPE* x1_outputFromGpu;
	DATA_TYPE* x2_outputFromGpu;
	DATA_TYPE* y_1;
	DATA_TYPE* y_2;

	a = (DATA_TYPE*)malloc(N*N*sizeof(DATA_TYPE));
	x1 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	x2 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	x1_outputFromGpu = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	x2_outputFromGpu = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	y_1 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	y_2 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));

	init_array(a, x1, x2, y_1, y_2);
	
	GPU_argv_init();

	mvtCuda(a, x1, x2, y_1, y_2, x1_outputFromGpu, x2_outputFromGpu);
	
	t_start = rtclock();

	//run the algorithm on the CPU
	runMvt(a, x1, x2, y_1, y_2);

	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(x1, x1_outputFromGpu, x2, x2_outputFromGpu);

	free(a);
	free(x1);
	free(x2);
	free(x1_outputFromGpu);
	free(x2_outputFromGpu);
	free(y_1);
	free(y_2);

  	return 0;
}

